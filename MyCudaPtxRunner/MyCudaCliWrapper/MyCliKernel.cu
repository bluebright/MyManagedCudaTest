#include "hip/hip_runtime.h"
﻿#include "MyCliKernel.cuh"

#include "hip/hip_runtime.h"
#include ""

__global__ void addKernel(int *c, int const* a, int const* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

