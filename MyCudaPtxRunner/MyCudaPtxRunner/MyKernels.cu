
#include <hip/hip_runtime.h>
﻿// note: the .cu suffix would be more typical, but the .c suffix
// makes it easier to setup in VS without requiring any configuration
// changes

typedef struct
{
	int Id;
	unsigned int Value;
} SomeBasicType;

extern "C"
{
	// the __global__ here makes it accessible as a callable kernel
	__global__ void Multiply(const int N, SomeBasicType* __restrict data, int factor)
	{
		// grid-stride loop
		for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
		{
			// note we want to mutate in place; we don't want to copy it out, update, copy back
			(data + i)->Value *= factor;
		}
	}
}
