/*
* Default code in Managedcuda document
*/

#include "hip/hip_runtime.h"


#include <stdio.h>


extern "C" {

	__global__ void multKernel(int *c, const int *a, const int *b)
	{
		int i = threadIdx.x;
		c[i] = a[i] * b[i];
	}

	int main() { return 0; }

}
